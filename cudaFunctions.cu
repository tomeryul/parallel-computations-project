#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void build_ReturndArr(double* data, double* matValues,int arrSize, double t) {

	int index = threadIdx.x;
	if(index < arrSize){
		double X = (data[index*5+2]-data[index*5+1])/2 * sin(t*M_PI/2) + (data[index*5+2]+data[index*5+1])/2;
		double Y = data[index*5+3]*X+data[index*5+4];
	
		matValues[index] = X;
		matValues[index+arrSize] = Y;
	
		
	}
}


double* GPUGetXY(double *data, int numElements, double t){

	hipError_t err = hipSuccess;
	
	// Allocate memory on GPU and CPU for the returnd Arr of X and Y
	size_t size_ReturndArrXY = (numElements/5)*2 * sizeof(double);
	double* h_myArr = (double*)malloc(size_ReturndArrXY);
	
	double *d_myArr = NULL;
    	err = hipMalloc((void **)&d_myArr, size_ReturndArrXY);
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to allocate returndArrXY memory - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
 
    	// Allocate memory on GPU to copy the data from the host
    	double *d_ArrData = NULL;
    	size_t size = numElements * sizeof(double);
    	err = hipMalloc((void **)&d_ArrData, size);
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to allocate device memory for d_ArrData- %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

    	// Copy data from host to the GPU memory
    	err = hipMemcpy(d_ArrData, data, size, hipMemcpyHostToDevice);
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
   	 }
   	 
   	  // Launch the Kernel
    	build_ReturndArr <<< 1 , PART >>> (d_ArrData, d_myArr, numElements/5, t);
    	
    	// Copy the  result of the d_myArr from GPU to the host memory.
    	err = hipMemcpy(h_myArr, d_myArr, size_ReturndArrXY, hipMemcpyDeviceToHost);
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to copy d_myArr to host - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    	
    	  err = hipGetLastError();
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to launch kernel function -  %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    	
    	 // Free allocated memory on GPU
    	if (hipFree(d_myArr) != hipSuccess) {
        	fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    	
     	// Free allocated memory on GPU
    	if (hipFree(d_ArrData) != hipSuccess) {
        	fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

    	
	
	return h_myArr; 
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


__global__ void init_HelpArr(int* helpArr) {

	int index = threadIdx.x;
	
	helpArr[index] = -1;
	
}

__global__ void build_HelpArr(int* d_helpArr, double* d_ArrValuesX, double* d_ArrValuesY, int sizeArr, double D, int K, int StartIndex) {

	
	int numClous = 0;
	double distanse;
	int index = threadIdx.x;
		
	int i = StartIndex*PART + index;


		
	if(i < sizeArr){
		for(int j = 0 ; j < sizeArr ; j++){
			if(i != j){
				distanse = sqrt(((d_ArrValuesX[j]-d_ArrValuesX[i])*(d_ArrValuesX[j]-d_ArrValuesX[i]))+((d_ArrValuesY[j]-d_ArrValuesY[i])*(d_ArrValuesY[j]-d_ArrValuesY[i])));
				
				if(distanse < D){
					numClous+=1;
					if(numClous == K){
						d_helpArr[index] = i;
						break;
					}
				}
			
			}
		}
	}
}


int* GPUGetPoints(double *allValuesX, double *allValuesY, int sizeArr, double D, int K, int StartIndex){

	hipError_t err = hipSuccess;
	int numPointsReturn = 0;

	
	int* h_returndPointsArr = (int*)malloc(3*sizeof(int));
	h_returndPointsArr[0] = -1;
	h_returndPointsArr[1] = -1;
	h_returndPointsArr[2] = -1;
	
	// Allocate memory on GPU and CPU for the helpArr
	// every point will have 1 spaces on the arr for 1 points thet is clous
	size_t size_helpArr = (PART) * sizeof(int);
	int* h_helpArr = (int*)malloc(size_helpArr);
	
	int *d_helpArr = NULL;
    	err = hipMalloc((void **)&d_helpArr, size_helpArr);
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to allocate device memory for d_helpArr - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    	
    	// Allocate memory on GPU to copy the allValuesX from the host
    	double *d_ArrValuesX = NULL;
    	size_t theSize = sizeArr * sizeof(double);
    	err = hipMalloc((void **)&d_ArrValuesX, theSize);
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to allocate device memory for d_ArrValuesX- %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

    	// Copy data from host to the GPU memory
    	err = hipMemcpy(d_ArrValuesX, allValuesX, theSize, hipMemcpyHostToDevice);
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
   	 }
    	
    	// Allocate memory on GPU to copy the allValuesY from the host
    	double *d_ArrValuesY = NULL;
    	err = hipMalloc((void **)&d_ArrValuesY, theSize);
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to allocate device memory for d_ArrValuesY- %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

    	// Copy data from host to the GPU memory
    	err = hipMemcpy(d_ArrValuesY, allValuesY, theSize, hipMemcpyHostToDevice);
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
   	 }
   	 
   	// Launch the Kernel
    	init_HelpArr <<< 1 , PART >>> (d_helpArr);
   	build_HelpArr <<< 1 , PART >>> (d_helpArr, d_ArrValuesX, d_ArrValuesY, sizeArr, D, K, StartIndex);
   	
   	
   	// Copy the  result of the d_helpArr from GPU to the host memory.
    	err = hipMemcpy(h_helpArr, d_helpArr, size_helpArr, hipMemcpyDeviceToHost);
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to copy d_helpArr to host - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    	
    	
    	for(int i = 0 ; i < PART ; i++){
    		if(h_helpArr[i]!= -1){
    			h_returndPointsArr[numPointsReturn] = h_helpArr[i];
    			numPointsReturn += 1;
    		}
    		if(numPointsReturn == 3)
    			break;
    	}
    	
    	err = hipGetLastError();
    	if (err != hipSuccess) {
        	fprintf(stderr, "Failed to launch kernel function -  %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    	
    	 // Free allocated memory on GPU
    	if (hipFree(d_helpArr) != hipSuccess) {
        	fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    	
     	// Free allocated memory on GPU
    	if (hipFree(d_ArrValuesX) != hipSuccess) {
        	fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    	
    	// Free allocated memory on GPU
    	if (hipFree(d_ArrValuesY) != hipSuccess) {
        	fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
    	

   	
	return h_returndPointsArr;

}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
